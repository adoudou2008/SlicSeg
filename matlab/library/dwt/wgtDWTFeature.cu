
#include <hip/hip_runtime.h>
// wgtDWTFeature
//
// This is a CUDA C++ file that is automatically compiled by the function CompileSlicSeg
//
// Author: Guotai Wang
// Copyright (c) 2014-2016 University College London, United Kingdom. All rights reserved.
// http://cmictig.cs.ucl.ac.uk
//
// Distributed under the BSD-3 licence. Please see the file licence.txt 
// This software is not certified for clinical use.
// 

__device__ double getPixel(const double *array,int H,int W,int i,int j)
{
    if(i<0 || i>=H || j<0 ||j>=W)
    {
        return 0;
    }
    else
    {
        return *(array+H*j+i);
    }
}
__device__ void setPixel(double *array,int H,int W,int i,int j,double value)
{
    if(i<0 || i>=H || j<0 ||j>=W)
    {
        return;
    }
    *(array+H*j+i)=value;
}

__global__ void wgtDWTFeature(const double * pDataLL2, const double * pDataLH2,
	const double * pDataHL2, const double * pDataHH2,
    const double * pDataLH1, const double * pDataHL1,
    const double * pDataHH1, double * FeatureMatrix, const int height, const int width)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if(i>=height || j>=width) return;

    int regionSize=8;
    int halfSize=8/2;
    int quaterSize=8/4;
    int featureLength=regionSize*regionSize;
    

    int starti=i-halfSize-1;
    int startj=j-halfSize-1;
    double tempValue=0;
    for(int x=0;x<regionSize;x++)
    {
        for(int y=0;y<regionSize;y++)
        {
            if(x<quaterSize && y<quaterSize)
            {
                tempValue=getPixel(pDataLL2, height, width, starti+4*x, startj+4*y);
            }
            else if(x>=quaterSize && x<halfSize && y<quaterSize)
            {
                tempValue=getPixel(pDataLH2, height, width, starti+4*(x-quaterSize), startj+4*y);
            }
            else if(x<quaterSize && y>=quaterSize && y<halfSize)
            {
                tempValue=getPixel(pDataHL2, height, width, starti+4*x, startj+4*(y-quaterSize));
            }
            else if(x>=quaterSize && x<halfSize && y>=quaterSize && y<halfSize)
            {
                tempValue=getPixel(pDataHH2, height, width, starti+4*(x-quaterSize), startj+4*(y-quaterSize));
            }
            else if(x>=halfSize && y<halfSize)
            {
                tempValue=getPixel(pDataLH1, height, width, starti+2*(x-halfSize), startj+2*y);
            }
            else if(x<halfSize && y>=halfSize)
            {
                tempValue=getPixel(pDataHL1, height, width, starti+2*x, startj+2*(y-halfSize));
            }
            else{
                tempValue=getPixel(pDataHH1, height, width, starti+2*(x-halfSize), startj+2*(y-halfSize));
            }
            setPixel(FeatureMatrix,height*width,featureLength,i+j*height,x+y*regionSize,tempValue);
        }
    }
}