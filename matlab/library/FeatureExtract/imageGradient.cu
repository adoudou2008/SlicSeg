
#include <hip/hip_runtime.h>
__device__ unsigned char getPixel(const unsigned char *array, int H, int W, int i, int j)
{
    if(i<0 || i>=H || j<0 ||j>=W)
    {
        return 0;
    }
    else
    {
        return *(array+H*j+i);
    }
}

__device__ void setPixel(double *array, int H, int W, int i, int j, double value)
{
    if(i<0 || i>=H || j<0 ||j>=W)
    {
        return;
    }
    *(array+H*j+i)=value;
}

__global__ void imageGradient(const unsigned char * inputData, double *g_mag, double *g_orient, const int H, const int W)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    double dx=getPixel(inputData, H, W, x+1, y)-getPixel(inputData, H, W, x-1, y);
    double dy=getPixel(inputData, H, W, x, y-1)-getPixel(inputData, H, W, x, y+1);
    double mag=sqrt(dx*dx+dy*dy);
    double orient=atan2(dy, dx);
    setPixel(g_mag, H, W, x, y, mag);
    setPixel(g_orient, H, W, x, y, orient);
}